#include "hip/hip_runtime.h"
/*
 *
 * CENG342 Project-3
 *
 * Downscaling SEQUENTIAL
 *
 * Usage:  main <input.jpg> <output.jpg> 
 *
 * @group_id 8
 * @author Emre Özçatal 20050111074, Semih Gür 19050111017, Emirhan Akıtürk 19050111065, Abdülsamet Haymana 19050111068
 *
 */
 
 #include <hip/hip_runtime.h>
#include <>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>


#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION

#include "stb_image.h"
#include "stb_image_write.h"

#define CHANNEL_NUM 4


/*
    Donnscaling function, simply sum the color of each 4 pixel and make it one pixel. 
*/
__global__ void downscaleImage(const uint8_t *inputImage, int width, int height, uint8_t *downsampledImage)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // To downscale it to half size 
    if (x < width / 2 && y < height / 2)
    {
        // Color defined in pixel as number up to 256
        int redSum = 0, greenSum = 0, blueSum = 0, alphaSum = 0;

        // Geting 4 pixel 
        for (int dy = 0; dy < 2; dy++)
        {
            for (int dx = 0; dx < 2; dx++)
            {
                int pixelIndex = ((y * 2 + dy) * width + (x * 2 + dx)) * CHANNEL_NUM;
                redSum += inputImage[pixelIndex];
                greenSum += inputImage[pixelIndex + 1];
                blueSum += inputImage[pixelIndex + 2];
                alphaSum += inputImage[pixelIndex + 3];
            }
        }

        // We will sum it to one pixel 
        int downsampledPixelIndex = (y * width / 2 + x) * CHANNEL_NUM;
        downsampledImage[downsampledPixelIndex] = redSum / 4;
        downsampledImage[downsampledPixelIndex + 1] = greenSum / 4;
        downsampledImage[downsampledPixelIndex + 2] = blueSum / 4;
        downsampledImage[downsampledPixelIndex + 3] = alphaSum / 4;
    }
}

void performCUDAImageDownscaling(const uint8_t *inputImage, int width, int height, uint8_t *downsampledImage)
{
    uint8_t *d_inputImage, *d_downsampledImage;
    size_t inputSize = width * height * CHANNEL_NUM * sizeof(uint8_t);
    size_t downsampledSize = (width / 2) * (height / 2) * CHANNEL_NUM * sizeof(uint8_t);

    // Allocate device memory
    hipMalloc((void **)&d_inputImage, inputSize);
    hipMalloc((void **)&d_downsampledImage, downsampledSize);

    // Copy input image data to device memory
    hipMemcpy(d_inputImage, inputImage, inputSize, hipMemcpyHostToDevice);

    // Set grid and block dimensions
    dim3 blockDim(16, 16);
    dim3 gridDim((width / 2 + blockDim.x - 1) / blockDim.x, (height / 2 + blockDim.y - 1) / blockDim.y);

    // Launch the kernel
    downscaleImage<<<gridDim, blockDim>>>(d_inputImage, width, height, d_downsampledImage);

    // Copy the result back to host memory
    hipMemcpy(downsampledImage, d_downsampledImage, downsampledSize, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_inputImage);
    hipFree(d_downsampledImage);
}

int main(int argc, char *argv[])
{
    clock_t start_time, end_time;
    double execution_time;

    int originalWidth, originalHeight, bpp;
    uint8_t *inputImage = stbi_load(argv[1], &originalWidth, &originalHeight, &bpp, CHANNEL_NUM);

    printf("Original Width: %d  Height: %d \n", originalWidth, originalHeight);
    printf("Input: %s , Output: %s  \n", argv[1], argv[2]);

    // Start measuring time
    start_time = clock(); 

    int downsampledWidth = originalWidth / 2;
    int downsampledHeight = originalHeight / 2;
    uint8_t *downsampledImage = (uint8_t *)malloc(downsampledWidth * downsampledHeight * CHANNEL_NUM * sizeof(uint8_t));

    performCUDAImageDownscaling(inputImage, originalWidth, originalHeight, downsampledImage);

    stbi_write_jpg(argv[2], downsampledWidth, downsampledHeight, CHANNEL_NUM, downsampledImage, 100);

    stbi_image_free(inputImage);
    free(downsampledImage);

    // Stop measuring time
    end_time = clock(); 

    execution_time = ((double)(end_time - start_time)) / CLOCKS_PER_SEC * 1000;

    printf("Execution time: %.2f milliseconds\n", execution_time);
    return 0;
}
